#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <unordered_map>
#include <map>
#include <chrono>
#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "../include/node.hpp"
#include "../include/tree.hpp"
#include "../include/gbm.hpp"
#include "../include/utils.hpp"


void GBM::train_hist_gpu(
		const std::vector<std::vector<uint8_t>>& _X_hist, 
		const std::vector<std::vector<uint8_t>>& _X_hist_rowmajor, 
		std::vector<float>& _y
		) {
	int n_rows = int(_y.size());

	thrust::device_vector<float> gradient(n_rows, 0.00f);
	thrust::device_vector<float> hessian(n_rows, 2.00f);

	// Add mean for better start.
	y_mean_train = get_vector_mean(_y);

	thrust::device_vector<float> round_preds(n_rows);
	thrust::device_vector<float> preds(n_rows, y_mean_train);

	const cuda_hist X_hist 			= convert_hist_to_cuda(_X_hist);
	const cuda_hist X_hist_rowmajor = convert_hist_to_cuda(_X_hist_rowmajor);
	const thrust::device_vector<float> y(_y);

	auto start_1 = std::chrono::high_resolution_clock::now();
	for (int round = 0; round < num_boosting_rounds; ++round) {
		trees.emplace_back(
					X_hist,
					gradient,
					hessian,
					max_depth,
					l2_reg,
					min_data_in_leaf,
					max_bin,
					max_leaves,
					col_subsample_rate,
					round
			);


		round_preds = trees[round].predict_hist_gpu(X_hist_rowmajor);
		for (int idx = 0; idx < n_rows; ++idx) {
			preds[idx] += lr * round_preds[idx];
		}

		gradient = calculate_gradient_gpu(preds, y);
		hessian  = calculate_hessian_gpu(preds, y);

		auto stop_1 	= std::chrono::high_resolution_clock::now();
		auto duration_1 = std::chrono::duration_cast<std::chrono::milliseconds>(stop_1 - start_1);

		if (round % verbosity == (verbosity - 1)) {
			printf("Round %i MSE Loss: %2.6f", round + 1, calculate_mse_loss_gpu(preds, y));
			printf("       ");
			printf("Num leaves: %i", (trees[round].num_leaves + 1) / 2);
			printf("       ");
			printf("Time Elapsed: %i ms", int(duration_1.count()));
		}
	}
}
