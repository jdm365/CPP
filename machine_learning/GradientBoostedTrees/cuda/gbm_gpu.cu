#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <unordered_map>
#include <map>
#include <chrono>
#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>
#include <thrust/execution_policy.h>

#include "../include/node.hpp"
#include "../include/tree.hpp"
#include "../include/gbm.hpp"
#include "../include/utils.hpp"


void GBM::train_hist_gpu(
		const std::vector<std::vector<uint8_t>>& _X_hist, 
		const std::vector<std::vector<uint8_t>>& X_hist_rowmajor, 
		std::vector<float>& y
		) {
	int n_rows = int(y.size());

	//thrust::device_vector<float> gradient(n_rows, 0.00f);
	thrust::device_vector<float> gradient;
	printf("made it");
	thrust::device_vector<float> hessian(n_rows, 2.00f);

	// Add mean for better start.
	y_mean_train = get_vector_mean(y);

	std::vector<float> round_preds(n_rows);
	std::vector<float> preds(n_rows, y_mean_train);

	const thrust::device_vector<uint8_t> X_hist = convert_hist_to_cuda(_X_hist);

	auto start_1 = std::chrono::high_resolution_clock::now();
	for (int round = 0; round < num_boosting_rounds; ++round) {
		trees.emplace_back(
					X_hist,
					gradient,
					hessian,
					max_depth,
					l2_reg,
					min_data_in_leaf,
					max_bin,
					max_leaves,
					col_subsample_rate,
					round
			);


		round_preds = trees[round].predict_hist(X_hist_rowmajor);
		for (int idx = 0; idx < n_rows; ++idx) {
			preds[idx] += lr * round_preds[idx];
		}

		gradient = calculate_gradient(preds, y);
		hessian  = calculate_hessian(preds, y);

		auto stop_1 	= std::chrono::high_resolution_clock::now();
		auto duration_1 = std::chrono::duration_cast<std::chrono::milliseconds>(stop_1 - start_1);

		if (round % verbosity == (verbosity - 1)) {
			printf("Round %i MSE Loss: %2.6f", round + 1, calculate_mse_loss(preds, y));
			printf("       ");
			printf("Num leaves: %i", (trees[round].num_leaves + 1) / 2);
			printf("       ");
			printf("Time Elapsed: %i ms", int(duration_1.count()));
		}
	}
}

/*
float GBM::calculate_mse_loss_gpu(
		thrust::device_vector<float>& preds, 
		const thrust::device_vector<float>& y
		) {
	thrust::device_vector<float> result(int(y.size()));

	thrust::transform(preds.begin(), preds.end(), y.begin(), result.begin(), thrust::minus<float>());

	float loss = 0.50f * thrust::inner_product(
			thrust::host, 
			result.begin(), 
			result.end(),
			result.begin(), 
			0.0f
			);
	loss /= float(y.size());
	return loss;
}
*/
