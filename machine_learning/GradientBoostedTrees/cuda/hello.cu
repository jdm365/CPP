#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>



#define N 1000000000

__global__ void vector_add(float* out, float* a, float* b, int n) {
	int thd_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (thd_idx < n) {
		out[thd_idx] = a[thd_idx] + b[thd_idx];
	}
}

int main() {
	float* a   = (float*)malloc(sizeof(float) * N);
	float* b   = (float*)malloc(sizeof(float) * N);
	float* out = (float*)malloc(sizeof(float) * N);

	float* d_a;
	float* d_b;
	float* d_out;

	for (int idx = 0; idx < N; ++idx) {
		a[idx] = 1.00f;
		b[idx] = 2.00f;
	}

	hipMalloc((void**)& d_a,   sizeof(float) * N);
	hipMalloc((void**)& d_b,   sizeof(float) * N);
	hipMalloc((void**)& d_out, sizeof(float) * N);

	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);


	int block_size = 256;
	int grid_size  = ((N + block_size) / block_size);

	vector_add<<<grid_size, block_size>>>(out, a, b, N);
	hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);
	

	free(a);
	free(b);
	free(out);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
	return 0;
}
