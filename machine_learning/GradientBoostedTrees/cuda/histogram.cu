#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <vector>


__global__ void vector_add(float* out, float* a, float* b, int n) {
	int thd_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (thd_idx < n) {
		out[thd_idx] = a[thd_idx] + b[thd_idx];
	}
}

int main() {
	std::vector<std::vector<uint8_t>> X_hist(100, std::vector<uint8_t>(100));

	// Create X_hist on gpu \\
	
	// X_hist is columnar
	int n_cols = int(X_hist.size());
	int n_rows = int(X_hist[0].size());

	uint8_t* X_hist_gpu;

	// Copy 2d std::vector to 1d array.
	uint8_t* X_hist_cpu = (uint8_t*)malloc(n_rows * n_cols);
	for (int col = 0; col < n_cols; ++col) {
		for (int row = 0; row < n_rows; ++row) {
			X_hist_cpu[col * n_rows + row] = X_hist[col][row];
		}
	}
	hipMalloc((void**)& X_hist_gpu, n_rows * n_cols);
	hipMemcpy(X_hist_gpu, X_hist_cpu, n_rows * n_cols, hipMemcpyHostToDevice);
	free(X_hist_cpu);


	int block_size = 256;
	int grid_size  = ((n_rows * n_cols + block_size) / block_size);

	vector_add<<<grid_size, block_size>>>(out, a, b, N);
	hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

	hipFree(X_hist_gpu);
	return 0;
}
