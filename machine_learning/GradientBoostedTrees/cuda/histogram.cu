#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "histogram.hpp"


template<typename T> void convert_cpu_array_to_gpu_array_2d(
		T** cpu_array,
		T** gpu_array,
		int n_rows,
		int n_cols
		) {

	// Copy 2d std::vector to array of col arrays.
	for (int col = 0; col < n_cols; ++col) {
		hipMalloc(&gpu_array[col], sizeof(T) * n_rows);
		hipMemcpy(
				gpu_array[col], 
				cpu_array[col], 
				sizeof(T) * n_rows, 
				hipMemcpyHostToDevice
				);
	}
	for (int col = 0; col < n_cols; ++col) {
		free(cpu_array[col]);
	}
	free(cpu_array);
}

template<typename T> void convert_cpu_array_to_gpu_array(
		T* cpu_array,
		T* gpu_array,
		int n_rows
		) {
	hipMalloc(&gpu_array, sizeof(T) * n_rows);
	std::cout << "test" << std::endl;
	
	hipMemcpy(
			gpu_array, 
			cpu_array, 
			sizeof(T) * n_rows, 
			hipMemcpyHostToDevice
			);
	free(cpu_array);
}

__global__ void calc_hists_single_feature(
		uint8_t* X_hist_gpu_col,
		Bin*   gpu_hist_col,
		float* ordered_gradients,
		float* ordered_hessians,
		int* row_idxs
		) {
	int thd_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int bin = (int)X_hist_gpu_col[row_idxs[thd_idx]];

	gpu_hist_col[bin].grad_sum += ordered_gradients[thd_idx];
	gpu_hist_col[bin].hess_sum += ordered_hessians[thd_idx];
	gpu_hist_col[bin].bin_cnt++;
}


void calc_hists_single_feature_wrapper(
		uint8_t** X_hist_cpu,
		Bin**   cpu_hist,
		float* ordered_gradients,
		float* ordered_hessians,
		int* row_idxs,
		int grid_size,
		int block_size,
		int n_rows,
		int n_cols
		) {
	float* ordered_gradients_gpu;
	convert_cpu_array_to_gpu_array(
			ordered_gradients,
			ordered_gradients_gpu,
			n_rows
			);


	float* ordered_hessians_gpu;
	convert_cpu_array_to_gpu_array(
			ordered_hessians,
			ordered_hessians_gpu,
			n_rows
			);


	int*   row_idxs_gpu;
	convert_cpu_array_to_gpu_array(
			row_idxs,
			row_idxs_gpu,
			n_rows
			);


	uint8_t** X_hist_gpu;
	convert_cpu_array_to_gpu_array_2d(
			X_hist_cpu, 
			X_hist_gpu, 
			n_rows, 
			n_cols
			);

	Bin** gpu_hist;
	convert_cpu_array_to_gpu_array_2d(
			cpu_hist, 
			gpu_hist, 
			n_rows, 
			n_cols
			);


	for (int col = 0; col < n_cols; ++col) {
		calc_hists_single_feature<<<grid_size, block_size>>>(
				X_hist_gpu[col], 
				gpu_hist[col],
				ordered_gradients, 
				ordered_hessians, 
				row_idxs
				);
	}
	hipFree(X_hist_gpu);
	hipFree(gpu_hist);

}

